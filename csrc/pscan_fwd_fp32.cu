#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "pscan_fwd_kernel.cuh"

template void pscan_fwd_cuda<float>(PScanParams &params, hipStream_t stream);
template void pscan_fwd_cuda<double>(PScanParams &params, hipStream_t stream);
